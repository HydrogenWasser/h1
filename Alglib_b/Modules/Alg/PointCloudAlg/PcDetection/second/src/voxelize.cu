#include "hip/hip_runtime.h"
//
// Created by root on 1/8/22.
//

#include "voxelize.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/types.h>
//#include <ATen/cuda/CUDAApplyUtils.cuh>

int const maxGridDim = 50000;

__device__ __forceinline__ static void reduceMax(float *address, float val) {
    int *address_as_i = reinterpret_cast<int *>(address);
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
                        __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old || __int_as_float(old) < val);
}

__device__ __forceinline__ static void reduceMax(double *address, double val) {
    unsigned long long *address_as_ull =
            reinterpret_cast<unsigned long long *>(address);
    unsigned long long old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(
                address_as_ull, assumed,
                __double_as_longlong(fmax(val, __longlong_as_double(assumed))));
    } while (assumed != old || __longlong_as_double(old) < val);
}

// get rid of meaningless warnings when compiling host code
#ifdef __CUDA_ARCH__

__device__ __forceinline__ static void reduceAdd(float *address, float val) {
#if (__CUDA_ARCH__ < 200)
#warning \
    "compute capability lower than 2.x. fall back to use CAS version of atomicAdd for float32"
    int *address_as_i = reinterpret_cast<int *>(address);
    int old = *address_as_i, assumed;
    do {
      assumed = old;
      old = atomicCAS(address_as_i, assumed,
                      __float_as_int(val + __int_as_float(assumed)));
    } while (assumed != old);
#else
    atomicAdd(address, val);
#endif
}

__device__ __forceinline__ static void reduceAdd(double *address, double val) {
#if (__CUDA_ARCH__ < 600)
#warning \
    "compute capability lower than 6.x. fall back to use CAS version of atomicAdd for float64"
    unsigned long long *address_as_ull =
        reinterpret_cast<unsigned long long *>(address);
    unsigned long long old = *address_as_ull, assumed;
    do {
      assumed = old;
      old = atomicCAS(address_as_ull, assumed,
                      __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
#else
    atomicAdd(address, val);
#endif
}
#endif



void dynamic_voxelize_gpu(const at::Tensor& points, at::Tensor& coors,
                          const std::vector<float> voxel_size,
                          const std::vector<float> coors_range,
                          const int NDim) {
    // check device
    CHECK_INPUT(points);

    at::cuda::CUDAGuard device_guard(points.device());

    const int num_points = points.size(0);
    const int num_features = points.size(1);

    const float voxel_x = voxel_size[0];
    const float voxel_y = voxel_size[1];
    const float voxel_z = voxel_size[2];
    const float coors_x_min = coors_range[0];
    const float coors_y_min = coors_range[1];
    const float coors_z_min = coors_range[2];
    const float coors_x_max = coors_range[3];
    const float coors_y_max = coors_range[4];
    const float coors_z_max = coors_range[5];

    const int grid_x = round((coors_x_max - coors_x_min) / voxel_x);
    const int grid_y = round((coors_y_max - coors_y_min) / voxel_y);
    const int grid_z = round((coors_z_max - coors_z_min) / voxel_z);

//    const int col_blocks = at::cuda::ATenCeilDiv(num_points, threadsPerBlock);
    const int col_blocks = ATenCeilDiv(num_points, threadsPerBlock);
    dim3 blocks(col_blocks);
    dim3 threads(threadsPerBlock);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

//    dynamic_voxelize_kernel<scalar_t, int><<<blocks, threads, 0, stream>>>(
//            points.contiguous().data_ptr<scalar_t>(),
//            coors.contiguous().data_ptr<int>(), voxel_x, voxel_y, voxel_z,
//            coors_x_min, coors_y_min, coors_z_min, coors_x_max, coors_y_max,
//            coors_z_max, grid_x, grid_y, grid_z, num_points, num_features, NDim);
    AT_DISPATCH_ALL_TYPES(points.scalar_type(), "dynamic_voxelize_kernel", [&] {
        dynamic_voxelize_kernel<scalar_t, int><<<blocks, threads, 0, stream>>>(
                points.contiguous().data_ptr<scalar_t>(),
                coors.contiguous().data_ptr<int>(), voxel_x, voxel_y, voxel_z,
                coors_x_min, coors_y_min, coors_z_min, coors_x_max, coors_y_max,
                coors_z_max, grid_x, grid_y, grid_z, num_points, num_features, NDim);
    });
    hipDeviceSynchronize();
    AT_CUDA_CHECK(hipGetLastError());

    return;
}

template<typename T>
__global__ void
feats_reduce_kernel(const T *feats, const int32_t *coors_map,
                    T *reduced_feats, // shall be 0 at initialization
                    const int num_input, const int num_feats,
                    const reduce_t reduce_type) {
    for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < num_input;
         x += gridDim.x * blockDim.x) {
        int32_t reduce_to = coors_map[x];
        if (reduce_to == -1) continue;

        const T *feats_offset = feats + x * num_feats;
        T *reduced_feats_offset = reduced_feats + reduce_to * num_feats;
        if (reduce_type == reduce_t::MAX) {
            for (int i = 0; i < num_feats; i++) {
                reduceMax(&reduced_feats_offset[i], feats_offset[i]);
            }
        } else {
            for (int i = 0; i < num_feats; i++) {
                reduceAdd(&reduced_feats_offset[i], feats_offset[i]);
            }
        }
    }
}

std::vector<at::Tensor> dynamic_point_to_voxel_forward_gpu(
        const at::Tensor &feats, const at::Tensor &coors,
        const reduce_t reduce_type) {
    CHECK_INPUT(feats);
    CHECK_INPUT(coors);

    const int num_input = feats.size(0);
    const int num_feats = feats.size(1);

    if (num_input == 0)
        return {feats.clone().detach(),
                coors.clone().detach(),
                coors.new_empty({0}, torch::kInt32),
                coors.new_empty({0}, torch::kInt32)};

    at::Tensor out_coors;
    at::Tensor coors_map;
    at::Tensor reduce_count;

    auto coors_clean = coors.masked_fill(coors.lt(0).any(-1, true), -1);

    std::tie(out_coors, coors_map, reduce_count) =
            at::unique_dim(coors_clean, 0, true, true, true);

    if (out_coors.index({0, 0}).lt(0).item<bool>()) {
        // the first element of out_coors (-1,-1,-1) and should be removed
        out_coors = out_coors.slice(0, 1);
        reduce_count = reduce_count.slice(0, 1);
        coors_map = coors_map - 1;
    }

    coors_map = coors_map.to(torch::kInt32);
    reduce_count = reduce_count.to(torch::kInt32);

    auto reduced_feats =
            at::empty({out_coors.size(0), num_feats}, feats.options());

    AT_DISPATCH_FLOATING_TYPES(
            feats.scalar_type(), "feats_reduce_kernel", ([&] {
        if (reduce_type == reduce_t::MAX)
            reduced_feats.fill_(-std::numeric_limits<scalar_t>::infinity());
        else
            reduced_feats.fill_(static_cast<scalar_t>(0));

        dim3 blocks(std::min(ATenCeilDiv(num_input, threadsPerBlock),
                             maxGridDim));
        dim3 threads(threadsPerBlock);
        feats_reduce_kernel<<<blocks, threads>>>(
                feats.data_ptr<scalar_t>(), coors_map.data_ptr<int32_t>(),
                reduced_feats.data_ptr<scalar_t>(), num_input, num_feats, reduce_type);
        if (reduce_type == reduce_t::MEAN)
            reduced_feats /= reduce_count.unsqueeze(-1).to(reduced_feats.dtype());
    }));
    AT_CUDA_CHECK(hipGetLastError());

    return {reduced_feats, out_coors};
}


torch::Tensor generate_voxel(torch::Tensor &points,
                             const std::vector<float> &voxel_size,
                             const std::vector<float> &coors_range,
                             const int NDim){
    torch::Tensor coors = points.new_zeros({points.size(0), 3},
                                           torch::TensorOptions().dtype(torch::kInt).device(torch::kCUDA, 0));
    dynamic_voxelize_gpu(points, coors, voxel_size, coors_range, 3);
    return coors;
}
std::vector<at::Tensor> DynamicSimpleVFE(torch::Tensor &voxels,
                                         torch::Tensor &coors){
    assert(coors.size(1) == 3 && voxels.size(0) == coors.size(0));
    std::vector<at::Tensor> res = dynamic_point_to_voxel_forward_gpu(voxels, coors, MEAN);
    assert(res.size() == 2);
    return res;
}


